#include "hip/hip_runtime.h"
#include "ToRORd_fkatp_endo.h"
#include <stddef.h>
#include <stdint.h>

__constant__  size_t pitch;
__constant__  real abstol;
__constant__  real reltol;
__constant__  real max_dt;
__constant__  real min_dt;
__constant__  uint8_t use_adpt;

size_t pitch_h;

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    uint8_t use_adpt_h = (uint8_t)solver->adaptive;

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(use_adpt), &use_adpt_h, sizeof(uint8_t)));
    log_to_stdout_and_file("Using ToRORd_fkatp_endo GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_h) {
        real reltol_h = solver->rel_tol;
        real abstol_h = solver->abs_tol;
        real max_dt_h = solver->max_dt;
        real min_dt_h = solver->min_dt;

        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(reltol), &reltol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(abstol), &abstol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(max_dt), &max_dt_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(min_dt), &min_dt_h, sizeof(real)));
        log_to_stdout_and_file("Using Adaptive Euler model to solve the ODEs\n");
    } else {
        log_to_stdout_and_file("Using Euler model to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_h)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes);

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();
    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(
            hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve,
                                    num_steps);

    check_cuda_error(hipPeekAtLastError());

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device)
        check_cuda_error(hipFree(cells_to_solve_device));
}

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadID < num_volumes) {

        *((real *)((char *)sv + pitch * 0) + threadID)  = -88.8691566357934;     // v millivolt                                 // v   --
        *((real *)((char *)sv + pitch * 1) + threadID)  = 0.0110752904836162;    // CaMKt millimolar                            // nai --
        *((real *)((char *)sv + pitch * 2) + threadID)  = 12.0996647655188;      // nai millimolar                              // nass --
        *((real *)((char *)sv + pitch * 3) + threadID)  = 12.1000028563765;      // nass millimolar                             // ki   --
        *((real *)((char *)sv + pitch * 4) + threadID)  = 142.412524737626;      // ki millimolar                               // kss  --
        *((real *)((char *)sv + pitch * 5) + threadID)  = 142.412481425842;      // kss millimolar                              // cai   --
        *((real *)((char *)sv + pitch * 6) + threadID)  = 7.45541572746214e-05;  // cai millimolar                              // cass   --
        *((real *)((char *)sv + pitch * 7) + threadID)  = 6.50418928341426e-05;  // cass millimolar                             // cansr  --
        *((real *)((char *)sv + pitch * 8) + threadID)  = 1.53037019085812;      // cansr millimolar                            // cajsr --
        *((real *)((char *)sv + pitch * 9) + threadID)  = 1.52803094224238;      // cajsr millimolar                            // m     -----
        *((real *)((char *)sv + pitch * 10) + threadID) = 0.000787657400526199;  // m dimensionless                             // hp   13
        *((real *)((char *)sv + pitch * 11) + threadID) = 0.830658198588696;     // h dimensionless                             // h  11
        *((real *)((char *)sv + pitch * 12) + threadID) = 0.830466744399495;     // j dimensionless                             // j   12
        *((real *)((char *)sv + pitch * 13) + threadID) = 0.674096901201792;     // hp dimensionless                            // jp
        *((real *)((char *)sv + pitch * 14) + threadID) = 0.830093612199637;    // jp dimensionless                            // mL
        *((real *)((char *)sv + pitch * 15) + threadID) = 0.000159670117055769; // mL dimensionless                            // hL
        *((real *)((char *)sv + pitch * 16) + threadID) = 0.528261721740178;    // hL dimensionless                            // hLp
        *((real *)((char *)sv + pitch * 17) + threadID) = 0.288775833197764;    // hLp dimensionless                           // a
        *((real *)((char *)sv + pitch * 18) + threadID) = 0.000944249645410894; // a dimensionless                             // iF
        *((real *)((char *)sv + pitch * 19) + threadID) = 0.999616956857814;    // iF dimensionless                            // iS
        *((real *)((char *)sv + pitch * 20) + threadID) = 0.593680589620082;    // iS dimensionless                            // ap
        *((real *)((char *)sv + pitch * 21) + threadID) = 0.000481107253796778; // ap dimensionless                            // iFp
        *((real *)((char *)sv + pitch * 22) + threadID) = 0.999616964658062;    // iFp dimensionless                           // iSp
        *((real *)((char *)sv + pitch * 23) + threadID) = 0.654092074678260;    // iSp dimensionless                           // d
        *((real *)((char *)sv + pitch * 24) + threadID) = 8.86091322819384e-29; // d dimensionless                             // ff
        *((real *)((char *)sv + pitch * 25) + threadID) = 0.999999992783113;    // ff dimensionless                            // fs
        *((real *)((char *)sv + pitch * 26) + threadID) = 0.938965241412012;    // fs dimensionless                            // fcaf
        *((real *)((char *)sv + pitch * 27) + threadID) = 0.999999992783179;    // fcaf dimensionless                          // fcas
        *((real *)((char *)sv + pitch * 28) + threadID) = 0.999900458262832;    // fcas dimensionless                          // jca---
        *((real *)((char *)sv + pitch * 29) + threadID) = 0.999977476316330;    // jca dimensionless                           // nca   32
        *((real *)((char *)sv + pitch * 30) + threadID) = 0.999999992566681;    // ffp dimensionless                           // nca_i 33
        *((real *)((char *)sv + pitch * 31) + threadID) = 0.999999992766279;    // fcafp dimensionless                         // ffp  30
        *((real *)((char *)sv + pitch * 32) + threadID) = 0.000492094765239740; // nca_ss dimensionless                        // fcafp  31
        *((real *)((char *)sv + pitch * 33) + threadID) = 0.000833711885764158; // nca_i dimensionless                         // xs1    39
        *((real *)((char *)sv + pitch * 34) + threadID) = 0.998073652444028;    // C3 dimensionless                            // xs2    40
        *((real *)((char *)sv + pitch * 35) + threadID) = 0.000844745297078649; // C2 dimensionless                            // Jrel_np  41
        *((real *)((char *)sv + pitch * 36) + threadID) = 0.000698171876592920; // C1 dimensionless                            // CaMKt   1
        *((real *)((char *)sv + pitch * 37) + threadID) = 0.000370404872169913; // O dimensionless                             // ikr_c0 34
        *((real *)((char *)sv + pitch * 38) + threadID) = 1.30239063420973e-05; // I dimensionless                             // ikr_c1 35
        *((real *)((char *)sv + pitch * 39) + threadID) = 0.247156543918935;    // xs1 dimensionless                           // ikr_c2 36
        *((real *)((char *)sv + pitch * 40) + threadID) = 0.000175017075236424; // xs2 dimensionless                           // ikr_o   37
        *((real *)((char *)sv + pitch * 41) + threadID) = 3.90843796133124e-24; // Jrel_np millimolar_per_millisecond          // ikr_i   38
        *((real *)((char *)sv + pitch * 42) + threadID) = -1.88428892080206e-22; // Jrel_p millimolar_per_millisecond           // Jrel_p   42

        if(use_adpt) {
            *((real *)((char *)sv + pitch * 43) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 44) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 45) + threadID) = 0.0;    // previous dt
        }
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve,
                          uint32_t num_cells_to_solve, int num_steps) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real *)((char *)sv + pitch * i) + sv_id) =
                        dt * rDY[i] + *((real *)((char *)sv + pitch * i) + sv_id);
                }
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], cur_time + max_dt, sv_id);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real final_time, int thread_id) {

    #define DT *((real *)((char *)sv + pitch * 43) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * 44) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * 45) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = powf(abstol, 2.0f);

    // dt = ((time_new + dt) > final_time) ? (final_time - time_new) : dt;
    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
    time_new += dt;

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

    int count = 0;

    int count_limit = (final_time - time_new) / min_dt;

    int aux_count_limit = count_limit + 2000000;

    if(aux_count_limit > 0) {
        count_limit = aux_count_limit;
    }

    while(1) {

        for(int i = 0; i < NEQ; i++) {
            // stores the old variables in a vector
            edos_old_aux_[i] = sv_local[i];
            // //computes euler method
            edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
            // steps ahead to compute the rk2 method
            sv_local[i] = edos_new_euler_[i];
        }

        time_new += dt;

        RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
        time_new -= dt; // step back

        real greatestError = 0.0, auxError = 0.0;
        //#pragma unroll
        for(int i = 0; i < NEQ; i++) {

            // stores the new evaluation
            _k2__[i] = rDY[i];
            _aux_tol = fabs(edos_new_euler_[i]) * reltol;
            _tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

            // finds the greatest error between  the steps
            auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

            greatestError = (auxError > greatestError) ? auxError : greatestError;
        }

        /// adapt the time step
        greatestError += __tiny_;
        previous_dt = dt;
        /// adapt the time step
        dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

        if(time_new + dt > final_time) {
            dt = final_time - time_new;
        }

        // it doesn't accept the solution
        if(count < count_limit && (greatestError >= 1.0f)) {
            // restore the old values to do it again
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_old_aux_[i];
            }
            count++;
            // throw the results away and compute again
        } else {
            count = 0;

            // if(greatestError >=1.0) {
            //    printf("Thread //d,accepting solution with error > //lf \n", threadID, greatestError);
            //}

            // it accepts the solutions
            // int aux = (dt > max_step && max_step != 0);
            // dt = (aux) ? max_step : dt;

            if(dt < min_dt) {
                dt = min_dt;
            }

            else if(dt > max_dt && max_dt != 0) {
                dt = max_dt;
            }

            if(time_new + dt > final_time) {
                dt = final_time - time_new;
            }

            // change vectors k1 e k2 , para que k2 seja aproveitado como k1 na proxima iteração
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                _k_aux__[i] = _k2__[i];
                _k2__[i] = _k1__[i];
                _k1__[i] = _k_aux__[i];
            }

            // it steps the method ahead, with euler solution
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_new_euler_[i];
            }

            // verifica se o incremento para a próxima iteração ultrapassa o tempo de salvar, q neste caso é o tempo
            // final
            if(time_new + previous_dt >= final_time) {
                // se são iguais, ja foi calculada a iteração no ultimo passo de tempo e deve-se para o laço
                // nao usar igualdade - usar esta conta, pode-se mudar a tolerância
                // printf("//d: //lf\n", threadID, fabs(final_time - time_new));
                if((fabs(final_time - time_new) < 1.0e-5)) {
                    break;
                } else if(time_new < final_time) {
                    dt = previous_dt = final_time - time_new;
                    time_new += previous_dt;
                    break;
                } else {
                    dt = previous_dt = min_dt;
                    time_new += (final_time - time_new);
                    printf("Nao era pra chegar aqui: %d: %lf\n", thread_id, final_time - time_new);
                    break;
                }
            } else {
                time_new += previous_dt;
            }
        }
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, int threadID_, real dt) {

    // State variables
    real v_old_;
    real CaMKt_old_;
    real nai_old_;
    real nass_old_;
    real ki_old_;
    real kss_old_;
    real cai_old_;
    real cass_old_;
    real cansr_old_;
    real cajsr_old_;
    real m_old_;
    real h_old_;
    real j_old_;
    real hp_old_;
    real jp_old_;
    real mL_old_;
    real hL_old_;
    real hLp_old_;
    real a_old_;
    real iF_old_;
    real iS_old_;
    real ap_old_;
    real iFp_old_;
    real iSp_old_;
    real d_old_;
    real ff_old_;
    real fs_old_;
    real fcaf_old_;
    real fcas_old_;
    real jca_old_;
    real ffp_old_;
    real fcafp_old_;
    real nca_ss_old_;
    real nca_i_old_;
    real C3_old_;
    real C2_old_;
    real C1_old_;
    real O_old_;
    real I_old_;
    real xs1_old_;
    real xs2_old_;
    real Jrel_np_old_;
    real Jrel_p_old_;

    if(use_adpt) {
        v_old_ = sv[0];
        CaMKt_old_ = sv[1];
        nai_old_ = sv[2];
        nass_old_ = sv[3];
        ki_old_ = sv[4];
        kss_old_ = sv[5];
        cai_old_ = sv[6];
        cass_old_ = sv[7];
        cansr_old_ = sv[8];
        cajsr_old_ = sv[9];
        m_old_ = sv[10];
        h_old_ = sv[11];
        j_old_ = sv[12];
        hp_old_ = sv[13];
        jp_old_ = sv[14];
        mL_old_ = sv[15];
        hL_old_ = sv[16];
        hLp_old_ = sv[17];
        a_old_ = sv[18];
        iF_old_ = sv[19];
        iS_old_ = sv[20];
        ap_old_ = sv[21];
        iFp_old_ = sv[22];
        iSp_old_ = sv[23];
        d_old_ = sv[24];
        ff_old_ = sv[25];
        fs_old_ = sv[26];
        fcaf_old_ = sv[27];
        fcas_old_ = sv[28];
        jca_old_ = sv[29];
        ffp_old_ = sv[30];
        fcafp_old_ = sv[31];
        nca_ss_old_ = sv[32];
        nca_i_old_ = sv[33];
        C3_old_ = sv[34];
        C2_old_ = sv[35];
        C1_old_ = sv[36];
        O_old_ = sv[37];
        I_old_ = sv[38];
        xs1_old_ = sv[39];
        xs2_old_ = sv[40];
        Jrel_np_old_ = sv[41];
        Jrel_p_old_ = sv[42];
    } else {
        //    //State variables
        v_old_ = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt_old_ = *((real *)((char *)sv + pitch * 1) + threadID_);
        nai_old_ = *((real *)((char *)sv + pitch * 2) + threadID_);
        nass_old_ = *((real *)((char *)sv + pitch * 3) + threadID_);
        ki_old_ = *((real *)((char *)sv + pitch * 4) + threadID_);
        kss_old_ = *((real *)((char *)sv + pitch * 5) + threadID_);
        cai_old_ = *((real *)((char *)sv + pitch * 6) + threadID_);
        cass_old_ = *((real *)((char *)sv + pitch * 7) + threadID_);
        cansr_old_ = *((real *)((char *)sv + pitch * 8) + threadID_);
        cajsr_old_ = *((real *)((char *)sv + pitch * 9) + threadID_);
        m_old_ = *((real *)((char *)sv + pitch * 10) + threadID_);
        h_old_ = *((real *)((char *)sv + pitch * 11) + threadID_);
        j_old_ = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp_old_ = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp_old_ = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL_old_ = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL_old_ = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp_old_ = *((real *)((char *)sv + pitch * 17) + threadID_);
        a_old_ = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF_old_ = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS_old_ = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap_old_ = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp_old_ = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp_old_ = *((real *)((char *)sv + pitch * 23) + threadID_);
        d_old_ = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff_old_ = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs_old_ = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf_old_ = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas_old_ = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca_old_ = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp_old_ = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp_old_ = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca_ss_old_ = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i_old_ = *((real *)((char *)sv + pitch * 33) + threadID_);
        C3_old_ = *((real *)((char *)sv + pitch * 34) + threadID_);
        C2_old_ = *((real *)((char *)sv + pitch * 35) + threadID_);
        C1_old_ = *((real *)((char *)sv + pitch * 36) + threadID_);
        O_old_ = *((real *)((char *)sv + pitch * 37) + threadID_);
        I_old_ = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1_old_ = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2_old_ = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np_old_ = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p_old_ = *((real *)((char *)sv + pitch * 42) + threadID_);
    }

    #include "ToROrd_common.inc.c"
}