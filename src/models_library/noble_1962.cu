#include "hip/hip_runtime.h"
#include "noble_1962.h"
#include <stddef.h>
#include <stdint.h>
#include "model_gpu_utils.h"

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    log_to_stdout_and_file("Using noble_1962 GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);

    // allocates a 2d contigous array
    check_cuda_error(hipMallocPitch((void **) &(solver->sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));


    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));


    //the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }
    solve_gpu <<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));

}

__global__ void kernel_set_model_inital_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {

        // Default values
        //*((real * )((char *) sv + pitch * 0) + threadID) = -75.5344986658f; //V millivolt 
        //*((real * )((char *) sv + pitch * 1) + threadID) = 0.060546727200f;   //m dimensionless 
        //*((real * )((char *) sv + pitch * 2) + threadID) = 0.725900135500f;   //h millivolt 
        //*((real * )((char *) sv + pitch * 3) + threadID) = 0.470923970800f;   //n dimensionless 

        // BCL = 300ms
        *((real * )((char *) sv + pitch * 0) + threadID) = -81.1893;    // V millivolt 
        *((real * )((char *) sv + pitch * 1) + threadID) = 0.0443563;    // m dimensionless
        *((real * )((char *) sv + pitch * 2) + threadID) = 0.851652;    // h dimensionless
        *((real * )((char *) sv + pitch * 3) + threadID) = 0.58291;    // n dimensionless
         
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id,dt);

            // Forward Euler
            *((real *) ((char *) sv + pitch * 0) + sv_id) = dt * rDY[0] + *((real *) ((char *) sv + pitch * 0) + sv_id);

            // Rush-Larsen
            *((real *) ((char *) sv + pitch * 1) + sv_id) = rDY[1];
            *((real *) ((char *) sv + pitch * 2) + sv_id) = rDY[2];
            *((real *) ((char *) sv + pitch * 3) + sv_id) = rDY[3];

        }

    }
}

inline __device__ void RHS_gpu(real *sv_, real *rDY_, real stim_current, int threadID_, real dt) {

    //State variables
    const real V_old_ =  *((real*)((char*)sv_ + pitch * 0) + threadID_);
    const real m_old_ =  *((real*)((char*)sv_ + pitch * 1) + threadID_);
    const real h_old_ =  *((real*)((char*)sv_ + pitch * 2) + threadID_);
    const real n_old_ =  *((real*)((char*)sv_ + pitch * 3) + threadID_);

    const real Cm = 12.0;               // (microF)
    const real g_na_max = 400.0;        // (microS)
    const real E_na = 40.0;             // (millivolt)
    const real g_L = 0.075;             // (microS)
    const real E_L = -60.0;             // (millivolt)

    real calc_I_stim = stim_current;

    // Algebraics
    real g_na =  powf(m_old_, 3.00000)*h_old_*g_na_max;
    real alpha_h = ((1.7e-01*exp((((-V_old_)-9.0e+01)/2.0e+01))));
    real alpha_m = (((1.0e-01*((-V_old_)-4.8e+01))/(exp((((-V_old_)-4.8e+01)/1.5e+01))-1.0e+00)));
    real alpha_n = (((1.0e-04*((-V_old_)-5.0e+01))/(exp((((-V_old_)-5.0e+01)/1.0e+01))-1.0e+00)));
    real i_na = (g_na+1.4e-01)*(V_old_ - E_na);
    //real i_na_no_oscilation = (g_na+1.2e-01)*(V_old_ - E_na);
    real beta_m = (((1.2e-01*(V_old_+8.0e+00))/(exp(((V_old_+8.0e+00)/5.0e+00))-1.0e+00)));
    real beta_h = ((1.0/(1.0e+00+exp((((-V_old_)-4.2e+01)/1.0e+01)))));
    real beta_n = ((2.0e-03*exp((((-V_old_)-9.0e+01)/8.0e+01))));
    real g_K1 = 1.2*exp((((-V_old_)-9.0e+01)/5.0e+01)) + (1.5e-02*exp(((V_old_+9.0e+01)/6.0e+01)));
    real g_K2 = 1.2*powf(n_old_,4.0e+00);
    real i_k =  (g_K1+g_K2)*(V_old_+100.000);
    real i_leak =  g_L*(V_old_ - E_L);

    real tau_h = 1.0 / (alpha_h + beta_h);
    real tau_m = 1.0 / (alpha_m + beta_m);
    real tau_n = 1.0 / (alpha_n + beta_n);
    real inf_h = alpha_h / (alpha_h + beta_h);
    real inf_m = alpha_m / (alpha_m + beta_m);
    real inf_n = alpha_n / (alpha_n + beta_n);

    // Rates
    rDY_[0] = (- (i_na + i_k + i_leak + calc_I_stim)/Cm);
    rDY_[1] = inf_m + (m_old_-inf_m)*expf(-dt/tau_m);
    rDY_[2] = inf_h + (h_old_-inf_h)*expf(-dt/tau_h);
    rDY_[3] = inf_n + (n_old_-inf_n)*expf(-dt/tau_n);

}

