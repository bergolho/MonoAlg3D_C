#include "hip/hip_runtime.h"
#include <stddef.h>
#include "../../monodomain/constants.h"
#include "../model_gpu_utils.h"

#include "tt2004_myo_epi.h"

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    log_to_stdout_and_file("Using mixed version of TT3 (MCELL + EPI) GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);
    size_t extra_data_bytes_size = num_volumes*sizeof(uint32_t);

    // TODO: Think what to do when the number of equations are different between the cellular models ...
    check_cuda_error(hipMallocPitch((void **) &(solver->sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    // Get the mapping array
    uint32_t *mapping = NULL;
    uint32_t *mapping_device = NULL;
    if(solver->ode_extra_data) {
        mapping = (uint32_t*)(solver->ode_extra_data);
        check_cuda_error(hipMalloc((void **)&mapping_device, extra_data_bytes_size));
        check_cuda_error(hipMemcpy(mapping_device, mapping, extra_data_bytes_size, hipMemcpyHostToDevice));
    }
    else {
        log_to_stderr_and_file_and_exit("You need to specify a mask function when using a mixed model!\n");
    }

    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(solver->sv, NULL, mapping_device, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    // Get the mapping array
    uint32_t *mapping = NULL, *mapping_device = NULL;
    if (ode_solver->ode_extra_data) {
        mapping = (uint32_t*)(ode_solver->ode_extra_data);
        check_cuda_error(hipMalloc((void **)&mapping_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(mapping_device, mapping, cells_to_solve_size, hipMemcpyHostToDevice));
    }
    else {
        log_to_stderr_and_file_and_exit("You need to specify a mask function when using a mixed model!\n");
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps, mapping_device);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if(mapping_device) check_cuda_error(hipFree(mapping_device));

}

__global__ void kernel_set_model_inital_conditions(real *sv, real *IC, uint32_t *mapping, int num_volumes) {

    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        
        // MCELL
        if (mapping[threadID] == 0) {
            *((real*)((char*)sv + pitch * 0) + threadID)  = INITIAL_V;   // V;       millivolt
            *((real*)((char*)sv + pitch * 1) + threadID)  = 0.f;         //M
            *((real*)((char*)sv + pitch * 2) + threadID)  = 0.75;    //H
            *((real*)((char*)sv + pitch * 3) + threadID)  = 0.75f;    //J
            *((real*)((char*)sv + pitch * 4) + threadID)  = 0.f;   //Xr1
            *((real*)((char*)sv + pitch * 5) + threadID)  = 1.f;    //Xr2
            *((real*)((char*)sv + pitch * 6) + threadID)  = 0.f;    //Xs
            *((real*)((char*)sv + pitch * 7) + threadID)  = 1.f;  //S
            *((real*)((char*)sv + pitch * 8) + threadID)  = 0.f;    //R
            *((real*)((char*)sv + pitch * 9) + threadID)  = 0.f;    //D
            *((real*)((char*)sv + pitch * 10) + threadID) = 1.f;   //F
            *((real*)((char*)sv + pitch * 11) + threadID) = 1.f; //FCa
            *((real*)((char*)sv + pitch * 12) + threadID) = 1.f;  //G
            *((real*)((char*)sv + pitch * 13) + threadID) = 0.0002;  //Cai
            *((real*)((char*)sv + pitch * 14) + threadID) = 0.2f;      //CaSR
            *((real*)((char*)sv + pitch * 15) + threadID) = 11.6f;   //Nai
            *((real*)((char*)sv + pitch * 16) + threadID) = 138.3f;    //Ki
        }
        // EPI
        else if (mapping[threadID] == 1) {
            *((real*)((char*)sv + pitch * 0) + threadID)  = INITIAL_V;   // V;       millivolt
            *((real*)((char*)sv + pitch * 1) + threadID)  = 0.f;   //M
            *((real*)((char*)sv + pitch * 2) + threadID)  = 0.75;    //H
            *((real*)((char*)sv + pitch * 3) + threadID)  = 0.75f;    //J
            *((real*)((char*)sv + pitch * 4) + threadID)  = 0.f;   //Xr1
            *((real*)((char*)sv + pitch * 5) + threadID)  = 1.f;    //Xr2
            *((real*)((char*)sv + pitch * 6) + threadID)  = 0.f;    //Xs
            *((real*)((char*)sv + pitch * 7) + threadID)  = 1.f;  //S
            *((real*)((char*)sv + pitch * 8) + threadID)  = 0.f;    //R
            *((real*)((char*)sv + pitch * 9) + threadID)  = 0.f;    //D
            *((real*)((char*)sv + pitch * 10) + threadID) = 1.f;   //F
            *((real*)((char*)sv + pitch * 11) + threadID) = 1.f; //FCa
            *((real*)((char*)sv + pitch * 12) + threadID) = 1.f;  //G
            *((real*)((char*)sv + pitch * 13) + threadID) = 0.0002;  //Cai
            *((real*)((char*)sv + pitch * 14) + threadID) = 0.2f;      //CaSR
            *((real*)((char*)sv + pitch * 15) + threadID) = 11.6f;   //Nai
            *((real*)((char*)sv + pitch * 16) + threadID) = 138.3f;    //Ki
        }
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps,
                          uint32_t *mapping) {

    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell 
    if(threadID < num_cells_to_solve) {

        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            // MCELL
            if (mapping[sv_id] == 0) {

                RHS_gpu_myo(sv, rDY, stim_currents[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real*)((char*)sv + pitch * i) + sv_id) = rDY[i];
                }
            }
            // EPI
            else if (mapping[sv_id] == 1) {

                RHS_gpu_epi(sv, rDY, stim_currents[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real*)((char*)sv + pitch * i) + sv_id) = rDY[i];
                }
            }
        }
    }
}

inline __device__ void RHS_gpu_myo (real *sv, real *rDY_, real stim_current, int threadID_, real dt) {

    // State variables
    real svolt = *((real*)((char*)sv + pitch * 0) + threadID_);
    real sm    = *((real*)((char*)sv + pitch * 1) + threadID_);
    real sh    = *((real*)((char*)sv + pitch * 2) + threadID_);
    real sj    = *((real*)((char*)sv + pitch * 3) + threadID_);
    real sxr1  = *((real*)((char*)sv + pitch * 4) + threadID_);
    real sxr2  = *((real*)((char*)sv + pitch * 5) + threadID_);
    real sxs   = *((real*)((char*)sv + pitch * 6) + threadID_);
    real ss    = *((real*)((char*)sv + pitch * 7) + threadID_);
    real sr    = *((real*)((char*)sv + pitch * 8) + threadID_);
    real sd    = *((real*)((char*)sv + pitch * 9) + threadID_);
    real sf    = *((real*)((char*)sv + pitch * 10) + threadID_);
    real sfca  = *((real*)((char*)sv + pitch * 11) + threadID_);
    real sg    = *((real*)((char*)sv + pitch * 12) + threadID_);
    real Cai   = *((real*)((char*)sv + pitch * 13) + threadID_);
    real CaSR  = *((real*)((char*)sv + pitch * 14) + threadID_);
    real Nai   = *((real*)((char*)sv + pitch * 15) + threadID_);
    real Ki    = *((real*)((char*)sv + pitch * 16) + threadID_);

    // Specific MCELL parameters
    real Gks = 0.062;
    real Gto = 0.294;

    real R_INF=1./(1.+exp((20-svolt)/6.));
    real S_INF=1./(1.+exp((svolt+20)/5.));
    real TAU_R=9.5*exp(-(svolt+40.)*(svolt+40.)/1800.)+0.8;
    real TAU_S=85.*exp(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+exp((svolt-20.)/5.))+3.;

    #include "tt2004_common.inc"

}

inline __device__ void RHS_gpu_epi (real *sv, real *rDY_, real stim_current, int threadID_, real dt) {
    // State variables
    real svolt = *((real*)((char*)sv + pitch * 0) + threadID_);
    real sm    = *((real*)((char*)sv + pitch * 1) + threadID_);
    real sh    = *((real*)((char*)sv + pitch * 2) + threadID_);
    real sj    = *((real*)((char*)sv + pitch * 3) + threadID_);
    real sxr1  = *((real*)((char*)sv + pitch * 4) + threadID_);
    real sxr2  = *((real*)((char*)sv + pitch * 5) + threadID_);
    real sxs   = *((real*)((char*)sv + pitch * 6) + threadID_);
    real ss    = *((real*)((char*)sv + pitch * 7) + threadID_);
    real sr    = *((real*)((char*)sv + pitch * 8) + threadID_);
    real sd    = *((real*)((char*)sv + pitch * 9) + threadID_);
    real sf    = *((real*)((char*)sv + pitch * 10) + threadID_);
    real sfca  = *((real*)((char*)sv + pitch * 11) + threadID_);
    real sg    = *((real*)((char*)sv + pitch * 12) + threadID_);
    real Cai   = *((real*)((char*)sv + pitch * 13) + threadID_);
    real CaSR  = *((real*)((char*)sv + pitch * 14) + threadID_);
    real Nai   = *((real*)((char*)sv + pitch * 15) + threadID_);
    real Ki    = *((real*)((char*)sv + pitch * 16) + threadID_);

    // Specific EPI parameters
    real Gks = 0.245;
    real Gto = 0.294;

    real R_INF=1./(1.+exp((20-svolt)/6.));
    real S_INF=1./(1.+exp((svolt+20)/5.));
    real TAU_R=9.5*exp(-(svolt+40.)*(svolt+40.)/1800.)+0.8;
    real TAU_S=85.*exp(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+exp((svolt-20.)/5.))+3.;

    #include "tt2004_common.inc"
}
